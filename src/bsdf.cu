#include "hip/hip_runtime.h"
#include "bsdf.h"

__host__ __device__ float dielectricFresnel(float cosThetaI, float eta) {
    float sinThetaTSq = eta * eta * (1.0f - cosThetaI * cosThetaI);

    // Total internal reflection
    if (sinThetaTSq > 1.0)
        return 1.0;

    float cosThetaT = glm::sqrt(glm::max(1.0 - sinThetaTSq, 0.0));

    float rs = (eta * cosThetaT - cosThetaI) / (eta * cosThetaT + cosThetaI);
    float rp = (eta * cosThetaI - cosThetaT) / (eta * cosThetaI + cosThetaT);

    return 0.5f * (rs * rs + rp * rp);
}